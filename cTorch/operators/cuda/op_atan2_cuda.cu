#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include "cTorch/operators/cuda/op_list_cuda.h"
#include "cTorch/operators/cuda/util_cuda.h"

#define atan2f_block(in_1, in_2, out, i)                                       \
  do {                                                                         \
    out_ptr_d[i] = atan2f(in_ptr_2_d[i], in_ptr_1_d[i]);                       \
  } while (0)

#define atan2d_block(in_1, in_2, out, i)                                       \
  do {                                                                         \
    out_ptr_d[i] = atan2(in_ptr_2_d[i], in_ptr_1_d[i]);                        \
  } while (0)

_cth_declare_cuda_binary_kernel_generic(
    cth_atan2_cuda, atan2f_block, atan2d_block);

/**
 * @brief Calculate the arc tangent of the ratio of first and second input
 * arguments
 *
 * @param[CTHOperator] op operator
 *
 * @note CUDA onlly support float & double type
 *
 * Inputs & outputs:
 *   - # of input: 2
 *   - # of output: 1
 *   - Assume input & output have same types
 */
void op_atan2_cuda(CTHOperator *op) {
  FORCE_OP_INPUT_OUTPUT_TENSOR_NUM(op, 2, 1);
  CTHTensor *input_1 = cth_array_at(CTHTensor)(op->in_bound_tensors, 0);
  CTHTensor *input_2 = cth_array_at(CTHTensor)(op->in_bound_tensors, 1);
  CTHTensor *output = cth_array_at(CTHTensor)(op->out_bound_tensors, 0);
  CTH_TENSOR_DEVICE device = input_1->meta_info->device;

  _cth_cuda_binary_workflow(
      input_1->meta_info->data_type,
      input_1->values,
      input_2->values,
      output->values,
      input_1->meta_info->n_elements,
      CTH_CUDA_THREADS_PER_BLOCK,
      cth_atan2_cuda,
      device);
}

#ifdef __cplusplus
}
#endif
