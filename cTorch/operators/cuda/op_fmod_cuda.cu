#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include "cTorch/operators/cuda/op_list_cuda.h"
#include "cTorch/operators/cuda/util_cuda.h"

_cth_declare_cuda_binary_kernel(cth_fmod_cuda, fmodf, fmod);

/**
 * @brief Calculate the double-precision floating-point remainder of x / y
 *
 * @param[CTorchOperator] op operator
 *
 * @note CUDA onlly support float & double type
 *
 * Inputs & outputs:
 *   - # of input: 2
 *   - # of output: 1
 *   - Assume input & output have same types
 */
void op_fmod_cuda(CTorchOperator *op) {
  FORCE_OP_INPUT_OUTPUT_TENSOR_NUM(op, 2, 1);
  CTorchTensor *input_1 = array_at(CTorchTensor)(op->in_bound_tensors, 0);
  CTorchTensor *input_2 = array_at(CTorchTensor)(op->in_bound_tensors, 1);
  CTorchTensor *output = array_at(CTorchTensor)(op->out_bound_tensors, 0);
  CTH_TENSOR_DEVICE device = input_1->meta_info->device;

  _cth_cuda_binary_workflow(
      input_1->meta_info->data_type,
      input_1->values,
      input_2->values,
      output->values,
      input_1->meta_info->n_elements,
      CTH_CUDA_THREADS_PER_BLOCK,
      cth_fmod_cuda,
      device);
}

#ifdef __cplusplus
}
#endif
