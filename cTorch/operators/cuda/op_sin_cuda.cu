#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include "cTorch/operators/cuda/op_list_cuda.h"
#include "cTorch/operators/cuda/util_cuda.h"

_cth_declare_cuda_unary_kernel(cth_sin_cuda, sinf, sin);

/**
 * @brief Calculate the sine of the input argument
 *
 * @param[CTorchOperator] op operator
 *
 * @note CUDA onlly support float & double type
 *
 * Inputs & outputs:
 *   - # of input: 1
 *   - # of output: 1
 *   - Assume input & output have same types
 */
void op_sin_cuda(CTorchOperator *op) {
  FORCE_OP_INPUT_OUTPUT_TENSOR_NUM(op, 1, 1);
  CTorchTensor *input = array_at(CTorchTensor)(op->in_bound_tensors, 0);
  CTorchTensor *output = array_at(CTorchTensor)(op->out_bound_tensors, 0);
  CTH_TENSOR_DEVICE device = input->meta_info->device;

  _cth_cuda_unary_workflow(
      input->meta_info->data_type,
      input->values,
      output->values,
      input->meta_info->n_elements,
      CTH_CUDA_THREADS_PER_BLOCK,
      cth_sin_cuda,
      device);
}

#ifdef __cplusplus
}
#endif
