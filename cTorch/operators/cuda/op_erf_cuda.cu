#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include "cTorch/operators/cuda/op_list_cuda.h"
#include "cTorch/operators/cuda/util_cuda.h"

_cth_declare_cuda_unary_kernel(cth_erf_cuda, erff, erf);

/**
 * @brief Calculate the error function of the input argument
 *
 * @param[CTHOperator] op operator
 *
 * @note CUDA onlly support float & double type
 *
 * Inputs & outputs:
 *   - # of input: 1
 *   - # of output: 1
 *   - Assume input & output have same types
 */
void op_erf_cuda(CTHOperator *op) {
  FORCE_OP_INPUT_OUTPUT_TENSOR_NUM(op, 1, 1);
  CTHTensor *input = cth_array_at(CTHTensor)(op->in_bound_tensors, 0);
  CTHTensor *output = cth_array_at(CTHTensor)(op->out_bound_tensors, 0);
  CTH_TENSOR_DEVICE device = input->meta_info->device;

  _cth_cuda_unary_workflow(
      input->meta_info->data_type,
      input->values,
      output->values,
      input->meta_info->n_elements,
      CTH_CUDA_THREADS_PER_BLOCK,
      cth_erf_cuda,
      device);
}

#ifdef __cplusplus
}
#endif
