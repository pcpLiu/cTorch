#include "hip/hip_runtime.h"
// Copyright 2021 Zhonghao Liu
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef __cplusplus
extern "C" {
#endif

#include "cTorch/operators/cuda/op_list_cuda.h"
#include "cTorch/operators/cuda/util_cuda.h"

_cth_declare_cuda_unary_kernel(cth_floor_cuda, floorf, floor);

/**
 * @brief Calculate the largest integer less than or equal to x
 *
 * @param[CTHOperator] op operator
 *
 * @note CUDA onlly support float & double type
 *
 * Inputs & outputs:
 *   - # of input: 1
 *   - # of output: 1
 *   - Assume input & output have same types
 */
void op_floor_cuda(CTHOperator *op) {
  FORCE_OP_INPUT_OUTPUT_TENSOR_NUM(op, 1, 1);
  CTHTensor *input = cth_array_at(CTHTensor)(op->in_bound_tensors, 0);
  CTHTensor *output = cth_array_at(CTHTensor)(op->out_bound_tensors, 0);
  CTH_TENSOR_DEVICE device = input->meta_info->device;

  _cth_cuda_unary_workflow(
      input->meta_info->data_type,
      input->values,
      output->values,
      input->meta_info->n_elements,
      CTH_CUDA_THREADS_PER_BLOCK,
      cth_floor_cuda,
      device);
}

#ifdef __cplusplus
}
#endif
