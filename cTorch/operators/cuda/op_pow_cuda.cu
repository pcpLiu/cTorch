#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include "cTorch/operators/cuda/op_list_cuda.h"
#include "cTorch/operators/cuda/util_cuda.h"

_cth_declare_cuda_binary_kernel(cth_pow_cuda, powf, pow);

/**
 * @brief Calculate the value of first argument to the power of second argument
 *
 * @param[CTorchOperator] op operator
 *
 * @note CUDA onlly support float & double type
 *
 * Inputs & outputs:
 *   - # of input: 2
 *   - # of output: 1
 *   - Assume input & output have same types
 */
void op_pow_cuda(CTorchOperator *op) {
  FORCE_OP_INPUT_OUTPUT_TENSOR_NUM(op, 2, 1);
  CTorchTensor *input_1 = array_at(CTorchTensor)(op->in_bound_tensors, 0);
  CTorchTensor *input_2 = array_at(CTorchTensor)(op->in_bound_tensors, 1);
  CTorchTensor *output = array_at(CTorchTensor)(op->out_bound_tensors, 0);
  CTH_TENSOR_DEVICE device = input_1->meta_info->device;

  _cth_cuda_binary_workflow(
      input_1->meta_info->data_type,
      input_1->values,
      input_2->values,
      output->values,
      input_1->meta_info->n_elements,
      CTH_CUDA_THREADS_PER_BLOCK,
      cth_pow_cuda,
      device);
}

#ifdef __cplusplus
}
#endif
