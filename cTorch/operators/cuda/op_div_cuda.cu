#include "hip/hip_runtime.h"
// Copyright 2021 Zhonghao Liu
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef __cplusplus
extern "C" {
#endif

#include "cTorch/operators/cuda/op_list_cuda.h"
#include "cTorch/operators/cuda/util_cuda.h"

#define _cth_div_cuda_atomic(x, y) (x / y)

_cth_declare_cuda_binary_kernel(cth_div_cuda, fdividef, _cth_div_cuda_atomic);

/**
 * @brief Divide two floating point values
 *
 * @param[CTHOperator] op operator
 *
 * @note CUDA onlly support float & double type
 *
 * Inputs & outputs:
 *   - # of input: 2
 *   - # of output: 1
 *   - Assume input & output have same types
 */
void op_div_cuda(CTHOperator *op) {
  FORCE_OP_INPUT_OUTPUT_TENSOR_NUM(op, 2, 1);
  CTHTensor *input_1 = cth_array_at(CTHTensor)(op->in_bound_tensors, 0);
  CTHTensor *input_2 = cth_array_at(CTHTensor)(op->in_bound_tensors, 1);
  CTHTensor *output = cth_array_at(CTHTensor)(op->out_bound_tensors, 0);
  CTH_TENSOR_DEVICE device = input_1->meta_info->device;

  _cth_cuda_binary_workflow(
      input_1->meta_info->data_type,
      input_1->values,
      input_2->values,
      output->values,
      input_1->meta_info->n_elements,
      CTH_CUDA_THREADS_PER_BLOCK,
      cth_div_cuda,
      device);
}

#ifdef __cplusplus
}
#endif
