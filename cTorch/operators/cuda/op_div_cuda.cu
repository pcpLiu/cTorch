#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include "cTorch/operators/cuda/op_list_cuda.h"
#include "cTorch/operators/cuda/util_cuda.h"

#define _cth_div_cuda_atomic(x, y) (x / y)

_cth_declare_cuda_binary_kernel(cth_div_cuda, fdividef, _cth_div_cuda_atomic);

/**
 * @brief Divide two floating point values
 *
 * @param[CTorchOperator] op operator
 *
 * @note CUDA onlly support float & double type
 *
 * Inputs & outputs:
 *   - # of input: 2
 *   - # of output: 1
 *   - Assume input & output have same types
 */
void op_div_cuda(CTorchOperator *op) {
  FORCE_OP_INPUT_OUTPUT_TENSOR_NUM(op, 2, 1);
  CTorchTensor *input_1 = array_at(CTorchTensor)(op->in_bound_tensors, 0);
  CTorchTensor *input_2 = array_at(CTorchTensor)(op->in_bound_tensors, 1);
  CTorchTensor *output = array_at(CTorchTensor)(op->out_bound_tensors, 0);
  CTH_TENSOR_DEVICE device = input_1->meta_info->device;

  _cth_cuda_binary_workflow(
      input_1->meta_info->data_type,
      input_1->values,
      input_2->values,
      output->values,
      input_1->meta_info->n_elements,
      CTH_CUDA_THREADS_PER_BLOCK,
      cth_div_cuda,
      device);
}

#ifdef __cplusplus
}
#endif
